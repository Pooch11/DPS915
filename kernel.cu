#include "hip/hip_runtime.h"
#pragma once
#include "WordProcessor.h"
#include <iostream>
#include <string.h>
#include <string>
#include <stdlib.h>
#include <fstream>
#include <cstdlib>
#include <iostream>
#include <iomanip>
#include <iterator>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "" // intellisense on CUDA syntax
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#define Timing 1
#define Serial 0
#define DEBUG 0
#define CHECK if (errorStatus != hipSuccess) {std::cout << errorStatus << std::endl;}
typedef struct  {
	int length;
	char* text;
	char* translate;
}Word;

using namespace std;
const int ntpb = 32;

hipError_t errorStatus = hipGetLastError();

//Device Functions


//string copy for device
__device__ char * my_strcpy(char *dest, const char *src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
	return dest;
}

//string concatenate for device
__device__ char * my_strcat(char *dest, const char *src) {
	int i = 0;
	while (dest[i] != 0) i++;
	my_strcpy(dest + i, src);
	return dest;
}

//'blackbox' - some device function that would replace text given a position
__device__ void replace_char(char s, char replace , int posBeg, int posEnd) {
			s = replace;
		s++;
}


__global__ void wordSearch(char *pszData, int dataLength, char *pszTarget, int targetLen, int *pFound, Word WA)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	printf("value = %c, address = %p\n", *pszData, (void *)pszData);
	if (*pFound > idx){
		// only continue if an earlier instance hasn't already been found
		int fMatch = 1;
		//printf("value = %c, address = %p\n", *pszData, (void *)pszData);
		for (int i = 0; i < targetLen; i++){ // we need to look for the next character
			if (pszData[idx + i] != pszTarget[i]) 
				fMatch = 0;
			//replace_char(pszTarget[idx + i], 'a');
			//call to strReplace;
		}
		if (fMatch)
			atomicMin(pFound, idx);
	}
}

void matchingCPU(char *T, int n, char *P, short m,bool *result)
{
int k; //keep track of string length
	for (int x = 0; x < n; x++){
		k = 0;
		for (int i = 0; i < m; i++)
			if (T[x + i] == P[i]) //starting to match char by char?
			++k;
		if (k == (m - 1)) { //character match up to length of pattern
			result[x] = true; //true for this start index
		}
	}

}

__global__ void matchingGPU(const char Target[], const char *Pattern, const int textLen, const int pattLen, volatile bool *result)
{
	extern __shared__ bool blockresults[];
	unsigned int Idx = threadIdx.x +blockDim.x * blockIdx.x; // 1 * 1 +  Idx
	if (Idx < textLen){
		 int k = 0;
		for (int i = 0; i < pattLen; i++) 
			if (Target[Idx + i] == Pattern[i]) 
				__syncthreads();
				++k;
		if (k == (pattLen - 1)) {  //if length we traversed is = to pattlen	
			blockresults[Idx] = true; //record match in an index handled by each separate thread
			__syncthreads();
		}
		result[Idx] = blockresults[Idx];
	}

}//end of kernel


#if Timing
#include <chrono>
#include <ctime>
//reportTime function made by Chris Szalwinski
void reportTime(const char* msg, std::chrono::steady_clock::duration span) {
	auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(span);
	std::cout << msg << " took - " << ms.count() << " millisecs" << std::endl;
}
#endif



	//Helper function to display - do not use with large files
	void displayFile(fstream& x) {
		x.clear();
		x.seekg(0, ios::beg);
		string line;
		while (std::getline(x, line)) {
			std::cout << line << std::endl;
		}
		x.clear();
		x.seekg(0, ios::beg);
	}
	//Helper function to display number of matches
	void matchcounter(int numOfResults, bool* h_result) {
		unsigned int matches = 0;
		for (int n = 0; n < numOfResults; n++) {
			if (h_result[n] == true) {
				//std::cout << "Found Match" << std::endl;
				matches++;
			}
			else {
				//std::cout << "No match found" << std::endl;
			}
		}
		cout << "Matches: \n" << matches << endl;
}

	//MAIN
	int main(int argc, char *argv[]){
#if Timing
		std::chrono::steady_clock::time_point ts, te, tmems, tmeme, tsCPU, teCPU, tsGPU, teGPU; //timestart, timeend, timecopy, timeCPU, timeGPU
#endif
		WordProcessor WP;
		
		
		if (argc != 3) {
			std::cerr << "Not enough arguments" << std::endl;
			system("pause");
			exit(3);
		}
		if (argc == 3) {
			std::string _testsearch = argv[2];
			if (_testsearch.find(".txt") == string::npos) {
				WP = WordProcessor(_testsearch);
			}
			else {
				WP = WordProcessor(argv[2]);
			}
			//initialize some name and file variables to read and write from
			char* fname = argv[1];
			fstream fp(fname);
			ofstream _tempfp("out.txt"); // this will be the file we write to
			string checkLine;
			unsigned int noLines = 0;
			if (!fp) {
				std::cout << "Cannot open/read file " << fname << std::endl;
				std::cerr << "Could not open/read file" << endl;
				system("pause");
				exit(1); //could not read
			}
#if Timing
			ts = std::chrono::steady_clock::now();
#endif
			//find size of file for array
			fp.seekg(0, std::ios::end);    // go to the end
			 int dataLen = fp.tellg();
			fp.seekg(0, std::ios::beg); //go to beginning
			//declare host and device arrays
			
			char* h_inputLine = new char[dataLen];
			
			//fill inputLine as buffer
			fp.read(h_inputLine, dataLen);

			//our word or pattern
			int patternsize = 4 ;
			char* h_word = new char[patternsize];

			//DEBUG
			strcpy (h_word, "the");


			int numOfResults = ceil( dataLen - ceil(dataLen%(patternsize )));
			//store our results set all to false - none found
			bool *h_result = (bool *)malloc(numOfResults * sizeof(bool));
			memset(h_result, false, numOfResults *sizeof(bool));

			//holders for DEVICE - dennoted by d_
			char* d_inputLine;
			char* d_word;
			bool* d_result;

			//alloc input and output for later use.
			hipMalloc((void**)&d_inputLine, dataLen * sizeof(char)); // input line to pass change
			hipMalloc( (void**)&d_word, patternsize * sizeof(char) ); //input pattern to find
			hipMalloc((void**)&d_result, numOfResults * sizeof(bool)); //store  matches at which indexes
			CHECK

			hipMemset((void**)d_result, false, numOfResults * sizeof(bool));
			CHECK
#if Timing 
			tmems = std::chrono::steady_clock::now();
			//copy memory to work HOST to DEVICE
			hipMemcpy(d_inputLine, h_inputLine, dataLen * sizeof(char), hipMemcpyHostToDevice);
			tmeme = std::chrono::steady_clock::now();
			reportTime("Memcopy to GPU took ", tmeme - tmems);
#endif
			CHECK
				std::cout << "Matching on CPU: " << std::endl;
#if Timing
			tsCPU = std::chrono::steady_clock::now();
			matchingCPU(h_inputLine, dataLen, h_word, patternsize, h_result);
			teCPU = std::chrono::steady_clock::now();
			reportTime("Finding Words in file CPU took ", teCPU - tsCPU);
#endif
			matchcounter(numOfResults, h_result);

			std::cout << "Matching on GPU: " << std::endl;

			//Grid declaration
			int nb = (dataLen + ntpb - 1) / ntpb;
			int results = dataLen / 32 ;
			dim3 dGrid(nb, 1 );
			dim3 dBlock(nb, 1);
#if Timing tsGPU = std::chrono::steady_clock::now();
#endif		

			matchingGPU << < 1, 1024 >> > (d_inputLine, d_word, dataLen, patternsize, d_result);
			hipDeviceSynchronize();
			CHECK

#if Timing teGPU = std::chrono::steady_clock::now();
			reportTime("Finding Words in file GPU took ", teGPU - tsGPU);
#endif

			//copy back DEVICE to HOST
			hipMemcpy(h_result, d_result, numOfResults * sizeof(bool), hipMemcpyDeviceToHost);
			CHECK

			//outputs
			matchcounter(numOfResults, h_result);
			//cout << "InputLine: \n" << h_inputLine << endl;
			//cout << "Pattern: \n" << h_word << endl;


#if Serial
			bool _modification = false;
			while (std::getline(fp, checkLine)) {
			_modification = WP.lookup(checkLine);
				_tempfp << checkLine;
				if (_modification)
					noLines++;
			}
			std::cout << "Processed " << noLines << " number of lines" << std::endl;
#endif

			if (noLines >= 0) { //number of modified lines 
				fp.close();
				_tempfp.close();
				//remove(fname);
				rename("out.txt", fname);
#if Timing
				te = std::chrono::steady_clock::now();
				reportTime("Program total ", te - ts);
#endif
				
			}
			else { //no changes to file old serial method
				_tempfp.close();
				remove("out.txt");
				fp.close();

				delete h_inputLine;
				delete h_word;
				delete h_result;

				hipFree(d_inputLine);
				hipFree(d_word);
				hipFree(d_result);
				system("pause");
				exit(2); //we were not able to write to file - or no changes to file
			}

			//Delete and Free memory
			delete h_inputLine;
			delete h_word;
			delete h_result;

			hipFree(d_inputLine);
			hipFree(d_word);
			hipFree(d_result);
		}

		system("pause");
		hipDeviceReset();
		return 0;
	
}
